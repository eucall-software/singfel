#include "hip/hip_runtime.h"
//#include <thrust/host_vector.h>
//#include <thrust/device_vector.h>
#include <iostream>
#include "diffraction.cuh"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/hip_math_constants.h>

//#define THREADS_PER_BLOCK 200

/*
double cuda_func(double ang) {
	// H has storage for 4 integers
    thrust::host_vector<int> H(4);
	
    // H and D are automatically deleted when the function returns
    return ang;
}

int cuda_func1(int *raw_ptr, int N) {
	// wrap raw pointer with a device_ptr 
	thrust::device_ptr<int> dev_ptr(raw_ptr);
	// use device_ptr in thrust algorithms
	thrust::fill(dev_ptr, dev_ptr + N, (int) 9);
	// access device memory through device_ptr
	dev_ptr[0] = 1;
    return dev_ptr[2];
}
*/

__global__ void addVec(int *a, int *b, int *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void addMat(float *a, float *b, float *c) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void addCube(float *a, float *b, float *c) { // colume-wise access
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

__global__ void matAmp(float *F,float *Fim, int numPix){
	int index = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
	if (index<numPix){
		F[index]=F[index]*F[index]+Fim[index]*Fim[index];
	}
}

/*
__global__ void structureFactor(float *F, float *f, float *q, float *p, int numPix, int numAtoms){
	//int index = blockIdx.x;
	int index = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
	if (index < numPix) {
		float sf_real = 0;
		float sf_imag = 0;
		// p (Nx3)
		// q (py x px x 3)
		float map = 0;
		int f_ind = 0;
		for (int n = 0; n < numAtoms; n++) {
			map = 6.283185307F * (p[n]*q[index] + p[n+numAtoms]*q[index+numPix] + p[n+(2*numAtoms)]*q[index+(2*numPix)]);
			f_ind = index+(n*numPix);
			sf_real += f[f_ind] * cos(map);
			sf_imag += f[f_ind] * sin(map);
		}
		F[index] = sf_real * sf_real + sf_imag * sf_imag;
	}
}
*/
	
//structureFactor<<<dim3(0x6,1),dim3(0xB,1)>>>(d_F,d_f,d_q,d_p,d_i,numPix,chunkSize);
__global__ void structureFactor(float *Fre, float *Fim, float *f, float *q, \
                                float *p, int *i, int numPix, int chunkSize) {
	int index = ( (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x \
	            + blockIdx.x ) * blockDim.x + threadIdx.x;
	if (index<numPix){
		// F (py x px) [re & im]
		// f (py x px x numAtomTypes)
		// q (py x px x 3)
		// i (1 x chunkSize)
		// p (chunkSize x 3)
		float map = 0;
		int f_ind = 0;
		for (int n = 0; n < chunkSize; n++) {
			map = 6.283185307F * (p[n] * q[index] + p[n+chunkSize] \
			      * q[index+(numPix)] + p[n+(2*chunkSize)]*q[index+(2*numPix)]);
			f_ind = index + i[n]*numPix;
			Fre[index] += f[f_ind] * cos(map);
			Fim[index] += f[f_ind] * sin(map);
		}
	}
}

/*
__global__ void structureFactorChunk(float *sf_real, float *sf_imag, float *f, float *q, int *i, float *p, int numAtomTypes, int numPix, int chunkSize){
	int index = ((blockIdx.y*blockDim.y + threadIdx.y)*gridDim.x + blockIdx.x )*blockDim.x + threadIdx.x;
	if (index<numPix){
		// F (py x px)
		// f (py x px x numAtomTypes)
		// q (py x px x 3)
		// i (1 x chunkSize)
		// p (chunkSize x 3)
		float map = 0;
		int f_ind = 0;
		for (int n = 0; n < chunkSize; n++) {
			map = 6.283185307F * (p[n]*q[index] + p[n+chunkSize]*q[index+(numPix)] + p[n+(2*chunkSize)]*q[index+(2*numPix)]);
			f_ind = index + i[n]*numPix;
			sf_real[index] += f[f_ind] * cos(map);
			sf_imag[index] += f[f_ind] * sin(map);
		}
	}
}

__global__ void structureFactorChunkParallel(float *pad_real, float *pad_imag, float *f, float *q, int *i, float *p, int numAtomTypes, int numPix, int chunkSize){
	int pixelId = blockIdx.x + blockIdx.y * gridDim.x;
	int chunkId = threadIdx.x;
	int index = pixelId + chunkId * numPix;
	if (pixelId < numPix && chunkId < chunkSize) {
		// F (py x px)
		// f (py x px x numAtomTypes)
		// q (py x px x 3)
		// i (1 x chunkSize)
		// p (chunkSize x 3)
		float map = 6.283185307F * (p[chunkId]*q[pixelId] + p[chunkId+chunkSize]*q[pixelId+(numPix)] + p[chunkId+(2*chunkSize)]*q[pixelId+(2*numPix)]);
		int f_ind = pixelId + i[chunkId]*numPix;
		pad_real[index] = f[f_ind] * cos(map);
		pad_imag[index] = f[f_ind] * sin(map);
	}
}
*/

void random_ints(int* a, int N)
{
   int i;
   for (i = 0; i < N; ++i)
    a[i] = rand() % 100; // between 0 and 100
}

void cuda_funcVec(int *a, int *b, int *c, int N) {
  int *d_a, *d_b, *d_c;
  int size = N*sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  addVec<<<N,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

void cuda_funcMat(float *a, float *b, float *c, int H, int W) {
  float *d_a, *d_b, *d_c;
  int size = H*W*sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  addMat<<<H*W,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

void cuda_funcCube(float *a, float *b, float *c, int H, int W, int Z) {
  float *d_a, *d_b, *d_c;
  int size = H*W*Z*sizeof(int);
  // Allocate space for device copies of a, b, c
  hipMalloc((void **)&d_a, size);
  hipMalloc((void **)&d_b, size);
  hipMalloc((void **)&d_c, size);
  // Copy inputs to device
  hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);
  // Launch add() kernel on GPU
  addCube<<<H*W*Z,1>>>(d_a, d_b, d_c);
  // Copy result back to host
  hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
  // Cleanup
  hipFree(d_a); hipFree(d_b); hipFree(d_c);
}

/*
void cuda_structureFactor(float *F, float *f, float *q, float *p, int py, int px, int numAtoms) {
	float *d_f, *d_q, *d_p;
	float *d_F;//, *d_F_real, *d_F_imag;
	int size_f = py*px*numAtoms*sizeof(float);
	int size_q = py*px*3*sizeof(float);
	int size_p = numAtoms*3*sizeof(float);
	int size_F = py*px*sizeof(float);
	// Malloc
	//float *F_real = (float *)malloc(size_F);
	//float *F_imag = (float *)malloc(size_F);
	// Allocate space for device copies of a, b, c
  	hipMalloc((void **)&d_f, size_f);
  	hipMalloc((void **)&d_q, size_q);
  	hipMalloc((void **)&d_p, size_p);
	hipMalloc((void **)&d_F, size_F);
	//hipMalloc((void **)&d_F_imag, size_F);
	// Copy inputs to device
  	hipMemcpy(d_f, f, size_f, hipMemcpyHostToDevice);
  	hipMemcpy(d_q, q, size_q, hipMemcpyHostToDevice);
  	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
  	// Launch add() kernel on GPU
  	structureFactor<<<py*px,1>>>(d_F, d_f, d_q, d_p, py, px, numAtoms);
  	// Copy result back to host
  	hipMemcpy(F, d_F, size_F, hipMemcpyDeviceToHost);
  	//hipMemcpy(F_imag, d_F_imag, size_F, hipMemcpyDeviceToHost);
	// Cleanup
  	hipFree(d_f); hipFree(d_q); hipFree(d_p), hipFree(d_F); //hipFree(d_F_imag);
	//free(F_real); free(F_imag);
}
*/

/*
void cuda_structureFactor(float *F, float *f, float *q, float *p, int numPix, int numAtoms) {
	float *d_F, *d_f, *d_q, *d_p;
	int size_F = numPix*sizeof(float);
	int size_f = numPix*numAtoms*sizeof(float);
	int size_q = numPix*3*sizeof(float);
	int size_p = numAtoms*3*sizeof(float);
	// Allocate space for device copies of a, b, c
	hipMalloc((void **)&d_F, size_F);
  	hipMalloc((void **)&d_f, size_f);
  	hipMalloc((void **)&d_q, size_q);
  	hipMalloc((void **)&d_p, size_p);
	// Copy inputs to device
  	hipMemcpy(d_f, f, size_f, hipMemcpyHostToDevice);
  	hipMemcpy(d_q, q, size_q, hipMemcpyHostToDevice);
  	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
  	// Launch add() kernel on GPU
	dim3 threads_per_block(20,10); // Maximum number of threads per block
	dim3 number_of_blocks(20,10,1);
  	structureFactor<<<number_of_blocks,threads_per_block>>>(d_F, d_f, d_q, d_p, numPix, numAtoms);
	//hipDeviceSynchronize();
  	//structureFactor<<<py*px,1>>>(d_F, d_f, d_q, d_p, py, px, numAtoms);
  	// Copy result back to host
  	hipMemcpy(F, d_F, size_F, hipMemcpyDeviceToHost);
	// Cleanup
  	hipFree(d_F); hipFree(d_f); hipFree(d_q); hipFree(d_p);
}
*/

void cuda_structureFactor(float *F, float *f, float *q, float *p, int *i, \
                          int numPix, int numAtoms, int numAtomTypes, \
                          int deviceID){
	int device,numBlocks;
	size_t size_F,size_f,size_q,size_i,size_p;
	size_t globalMem,fixedMem;
	int chunk,chunkSize,chunkSizeMax;
	float *d_F,*d_f,*d_p,*d_q,*d_Fim;
	int *d_i;
	struct hipDeviceProp_t prop; 
	dim3 dimG,dimB;
	size_F = sizeof(float) * numPix;
	size_f = sizeof(float) * numPix * numAtomTypes;
	size_q = sizeof(float) * numPix * 3;
	fixedMem = size_F*2 + size_f + size_q;
	dimB.x = CUDA_BLOCK_SIZE;
	dimG.x = numBlocks = (numPix+dimB.x-1)/dimB.x;
	if (dimG.x > CUDA_GROUP_LIMIT){
		dimG.y = (numBlocks + CUDA_GROUP_LIMIT -1) / CUDA_GROUP_LIMIT;
		dimG.x = (numBlocks + dimG.y -1) / dimG.y;
	}
	//hipGetDevice(&device);
	device = deviceID;
	hipSetDevice(device);
	hipGetDeviceProperties(&prop,device);
	
	globalMem = prop.totalGlobalMem  - CUDA_RESERVE_MEM;
	if (globalMem <= fixedMem){
		printf("Device memory[%lu] not enough to hold all data[>%lu]!\n", \
		       globalMem,fixedMem);
		exit(EXIT_FAILURE);
	}	
	chunkSizeMax = (globalMem - fixedMem) / (3 * sizeof(float) + sizeof(int));
	if (chunkSizeMax>numAtoms)
		chunkSizeMax=numAtoms;

	size_p = sizeof(float) * chunkSizeMax * 3;
	size_i = sizeof(int)   * chunkSizeMax;

	hipMalloc((void **)&d_F, size_F);
	hipMalloc((void **)&d_Fim, size_F);
  	hipMalloc((void **)&d_f, size_f);
  	hipMalloc((void **)&d_q, size_q);
  	hipMalloc((void **)&d_p, size_p);
  	hipMalloc((void **)&d_i, size_i);
  	hipMemcpy(d_f, f, size_f, hipMemcpyHostToDevice);
  	hipMemcpy(d_q, q, size_q, hipMemcpyHostToDevice);
  	hipMemset(d_F,0,size_F);
  	hipMemset(d_Fim,0,size_F);
 	
  	chunkSize=chunkSizeMax;
  	for(chunk=0;chunk<numAtoms;chunk+=chunkSizeMax){
		
		if (chunkSize+chunk>=numAtoms)
			chunkSize=numAtoms-chunk;
		hipMemcpy(d_i, i + chunk, sizeof(int)*	chunkSize, \
		           hipMemcpyHostToDevice);
		hipMemcpy(d_p, p + chunk, sizeof(float)*chunkSize, \
		           hipMemcpyHostToDevice);
		hipMemcpy(d_p + chunkSize, p + chunk + numAtoms, \
		           sizeof(float)*chunkSize, hipMemcpyHostToDevice);
		hipMemcpy(d_p + 2*chunkSize, p + chunk + 2*numAtoms, \
		           sizeof(float)*chunkSize, hipMemcpyHostToDevice);
		structureFactor<<<dimG,dimB>>>(d_F, d_Fim, d_f, d_q, d_p, d_i, numPix, \
		                               chunkSize);
	}
  	matAmp<<<dimG,dimB>>>(d_F,d_Fim,numPix);
  	hipMemcpy(F, d_F, size_F, hipMemcpyDeviceToHost);
  	hipFree(d_F);
  	hipFree(d_Fim);
  	hipFree(d_f);
  	hipFree(d_q);
  	hipFree(d_p);
  	hipFree(d_i);
}

int cuda_getDeviceCount(){
	int tmp;
	if (hipSuccess != hipGetDeviceCount(&tmp))
		return 0;
	return tmp;
}

/*	
void cuda_structureFactorChunk(float *sf_real, float *sf_imag, float *f, float *q, int *i, float *p, int numAtomTypes, int numPix, int chunkSize) {
	float *d_sf_real, *d_sf_imag, *d_f, *d_q, *d_p; // Pointer to device memory
	int *d_i;
	int size_sf = numPix*sizeof(float);
	int size_f = numPix*numAtomTypes*sizeof(float);
	int size_q = numPix*3*sizeof(float);
	int size_i = chunkSize*sizeof(int);
	int size_p = chunkSize*3*sizeof(float);
	// Allocate space for device copies
	hipMalloc((void **)&d_sf_real, size_sf);
	hipMalloc((void **)&d_sf_imag, size_sf);
  	hipMalloc((void **)&d_f, size_f);
  	hipMalloc((void **)&d_q, size_q);
	hipMalloc((void **)&d_i, size_i);
  	hipMalloc((void **)&d_p, size_p);
	// Copy inputs to device
  	hipMemcpy(d_sf_real, sf_real, size_sf, hipMemcpyHostToDevice);
  	hipMemcpy(d_sf_imag, sf_imag, size_sf, hipMemcpyHostToDevice);
  	hipMemcpy(d_f, f, size_f, hipMemcpyHostToDevice);
  	hipMemcpy(d_q, q, size_q, hipMemcpyHostToDevice);
  	hipMemcpy(d_i, i, size_i, hipMemcpyHostToDevice);
  	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
  	// Launch kernel on GPU
	dim3 threads_per_block(512); // Maximum number of threads per block
	dim3 number_of_blocks(2048,2048,1);
  	//structureFactorChunk<<<number_of_blocks,threads_per_block>>>(d_sf_real, d_sf_imag, d_f, d_q, d_i, d_p, numAtomTypes, numPix, chunkSize);
	structureFactorChunkParallel<<<number_of_blocks,threads_per_block>>>(d_sf_real, d_sf_imag, d_f, d_q, d_i, d_p, numAtomTypes, numPix, chunkSize);
	//hipDeviceSynchronize();
  	// Copy result back to host
  	hipMemcpy(sf_real, d_sf_real, size_sf, hipMemcpyDeviceToHost);
  	hipMemcpy(sf_imag, d_sf_imag, size_sf, hipMemcpyDeviceToHost);
	// Cleanup
  	hipFree(d_sf_real); hipFree(d_sf_imag); hipFree(d_f); hipFree(d_q); hipFree(d_i); hipFree(d_p);
}

void cuda_structureFactorChunkParallel(float *pad_real, float *pad_imag, float *f, float *q, int *i, float *p, int numAtomTypes, int numPix, int chunkSize) {
	float *d_pad_real, *d_pad_imag, *d_f, *d_q, *d_p; // Pointer to device memory
	int *d_i;
	int size_pad = numPix*chunkSize*sizeof(float);
	int size_f = numPix*numAtomTypes*sizeof(float);
	int size_q = numPix*3*sizeof(float);
	int size_i = chunkSize*sizeof(int);
	int size_p = chunkSize*3*sizeof(float);
	// Allocate space for device copies
  	hipMalloc((void **)&d_pad_real, size_pad);
  	hipMalloc((void **)&d_pad_imag, size_pad);
  	hipMalloc((void **)&d_f, size_f);
  	hipMalloc((void **)&d_q, size_q);
	hipMalloc((void **)&d_i, size_i);
  	hipMalloc((void **)&d_p, size_p);
	// Copy inputs to device
  	hipMemcpy(d_f, f, size_f, hipMemcpyHostToDevice);
  	hipMemcpy(d_q, q, size_q, hipMemcpyHostToDevice);
  	hipMemcpy(d_i, i, size_i, hipMemcpyHostToDevice);
  	hipMemcpy(d_p, p, size_p, hipMemcpyHostToDevice);
  	// Launch kernel on GPU
	dim3 threads_per_block(chunkSize); // Maximum number of threads per block
	dim3 number_of_blocks(numPix);
	structureFactorChunkParallel<<<number_of_blocks,threads_per_block>>>(d_pad_real, d_pad_imag, d_f, d_q, d_i, d_p, numAtomTypes, numPix, chunkSize);
  	// Copy result back to host
  	hipMemcpy(pad_real, d_pad_real, size_pad, hipMemcpyDeviceToHost);
  	hipMemcpy(pad_imag, d_pad_imag, size_pad, hipMemcpyDeviceToHost);
	// Cleanup
  	hipFree(d_pad_real); hipFree(d_pad_imag); hipFree(d_f); hipFree(d_q); hipFree(d_i); hipFree(d_p);
}
*/

